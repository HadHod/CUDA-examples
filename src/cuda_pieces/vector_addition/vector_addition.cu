#include "vector_addition.cuh"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__device__ void
vectorAddition_kernel(int* out, int* in) {
    int i = threadIdx.x;

    if (i == 1) {
        out = in;
    }
}

void AddVectors() {
    int* dev_a;
    int* dev_b;
    int a = 1;
    int b = 0;

    hipMalloc((void**) &dev_a, sizeof(int));
    hipMalloc((void**) &dev_b, sizeof(int));

    hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice);

    const dim3 gridDim(1, 1, 1);
    const dim3 blockDim(1, 1, 1);

    vectorAddition_kernel<<<gridDim, blockDim>>>(dev_b, dev_a);
    hipDeviceSynchronize();

    hipMemcpy(b, &dev_b, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Result: " << (b == 1) ? "ok\n" : "wrong\n";

    hipFree(dev_a);
    hipFree(dev_b);
}
