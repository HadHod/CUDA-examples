#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "vector_addition.cuh"
#include "kernel/main_kernel.cuh"

void AddVectors() {
    const dim3 gridDim(1, 1, 1);
    const dim3 blockDim(1, 1, 1);

    vectorAddition_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
}

void AddVectors(int* a, int* b) {

}
