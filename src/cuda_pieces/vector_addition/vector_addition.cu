#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "vector_addition.cuh"
#include "kernel/main_kernel.cuh"
#include "../../utils/error_checking.cuh"

void AddVectors() {

}

void AddVectors(const int* a, const int* b, const int dataLength) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    const int threadsPerBlock = min(prop.maxThreadsPerBlock, prop.maxThreadsDim[0]); // Number of threads per block can be less then max threads dimension?

    const dim3 gridDim(ceil((float) dataLength / threadsPerBlock), 1, 1);
    const dim3 blockDim(threadsPerBlock, 1, 1);

    vectorAddition_kernel<<<gridDim, blockDim>>>();
    gpuErrorCheck( hipDeviceSynchronize() );
}
