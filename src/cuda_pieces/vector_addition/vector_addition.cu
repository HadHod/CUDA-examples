#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "vector_addition.cuh"
#include "kernel/main_kernel.cuh"
#include "../../utils/error_checking.cuh"

void AddVectors() {
    const dim3 gridDim(1, 1, 1);
    const dim3 blockDim(1, 1, 1);

    vectorAddition_kernel<<<1, 1>>>();
    gpuErrorCheck( hipDeviceSynchronize() );
}

void AddVectors(int* a, int* b) {

}
