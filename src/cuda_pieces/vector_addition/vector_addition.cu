#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "vector_addition.cuh"
#include "kernel/main_kernel.cuh"
#include "../../utils/error_checking.cuh"

using namespace std;

void AddVectors() {
    // TODO
}

void AddVectors(const int* a, const int* b, const int dataLength) {
    const size_t MEMORY_SIZE = dataLength * sizeof(int);

    int* result = new int[dataLength];

    int* dev_a;
    int* dev_b;
    int* dev_result;

    gpuErrorCheck( hipMalloc((void**) &dev_a, MEMORY_SIZE) );
    gpuErrorCheck( hipMalloc((void**) &dev_b, MEMORY_SIZE) );
    gpuErrorCheck( hipMalloc((void**) &dev_result, MEMORY_SIZE) );

    gpuErrorCheck( hipMemcpy(dev_a, a, MEMORY_SIZE, hipMemcpyHostToDevice) );
    gpuErrorCheck( hipMemcpy(dev_b, b, MEMORY_SIZE, hipMemcpyHostToDevice) );

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    const int threadsPerBlock = min(prop.maxThreadsPerBlock, prop.maxThreadsDim[0]); // Number of threads per block can be less then max threads dimension?

    const dim3 gridDim((dataLength / threadsPerBlock) + 1, 1, 1);
    const dim3 blockDim(threadsPerBlock, 1, 1);

    vectorAddition_kernel<<<gridDim, blockDim>>>(dev_result, dev_a, dev_b, dataLength);
    gpuErrorCheck( hipPeekAtLastError() );
    gpuErrorCheck( hipDeviceSynchronize() );

    gpuErrorCheck( hipMemcpy(result, dev_result, MEMORY_SIZE, hipMemcpyDeviceToHost) );

    for (int i=0; i<10; i++) {
        cout << i << " : " << result[i] << " = " << a[i] << " + " << b[i] << "\n";
    }

    delete[] a;
    delete[] b;
    delete[] result;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_result);
}
