#include "hip/hip_runtime.h"
#include "main_kernel.cuh"

__global__ void
vectorAddition_kernel(int* result, const int* a, const int* b, const int size) {
    unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < size) {
        result[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}
