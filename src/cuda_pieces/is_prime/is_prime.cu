#include "is_prime.cuh"
#include "kernel/main_kernel.cuh"

#include <iostream>

using namespace std;

void IsPrime(int a) {
    int* number = &a;
    int* dev_number;
    bool result = false;
    bool* dev_result;

    hipMalloc((void**) &dev_result, sizeof(bool));
    hipMalloc((void**) &dev_number, sizeof(int));

    hipMemcpy(dev_number, number, sizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(1, 1, 1);
    isPrime_kernel<<<dimGrid, dimBlock>>>(*dev_result);
    hipDeviceSynchronize();

    hipMemcpy(&result, dev_result, sizeof(bool), hipMemcpyDeviceToHost);

    hipFree(dev_result);
    hipFree(dev_number);

    cout << "Result: " << result << "\n";
}
