#include "is_prime.cuh"
#include "kernel/main_kernel.cuh"

#include <iostream>

using namespace std;

void IsPrime(int a) {
    int* number = &a;
    int* dev_number;
    bool* result = malloc(sizeof(bool));
    bool* dev_result;

    hipMalloc((void**) &dev_result, sizeof(bool));
    hipMalloc((void**) &dev_number, sizeof(int));

    hipMemcpy(dev_number, number, sizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(1, 1, 1);
    isPrime_kernel<<<dimGrid, dimBlock>>>(result, number);
    hipDeviceSynchronize();

    hipMemcpy(result, dev_result, sizeof(bool), hipMemcpyDeviceToHost);

    hipFree(dev_result);
    hipFree(dev_number);

    cout << "Result: " << result << "\n";
}
