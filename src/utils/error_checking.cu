#include "error_checking.cuh"

#include <iostream>

inline void gpuAssert(const hipError_t code, const char *file, int line, const bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) {
            exit(code);
        }
    }
}

void gpuErrorCheck(hipError_t ans) {
    gpuAssert((ans), __FILE__, __LINE__);
}
